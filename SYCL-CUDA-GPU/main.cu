#include "hip/hip_runtime.h"
#include "lbvh.cuh"
#include <random>
#include <vector>
#include <chrono>
#include <thrust/random.h>
#include <tbb/parallel_invoke.h>
#define TBB_PREVIEW_GLOBAL_CONTROL 1

#include "tbb/global_control.h"
// #include <hipcub/hipcub.hpp>
// #include <cub/device/device_radix_sort.cuh>

using real_t = double;
using real_s = double4;
using node_t = lbvh::detail::node;
using aabb_t = lbvh::aabb<real_t>;

#define TOL 1e-10
#define FW_S32_MIN  (~0x7FFFFFFF)

struct aabb_getter
{
  /* aqui destaca que el limite superior e inferior que le da a las cajas es el mismo,
  es decir, la caja es un solo punto */
  __device__
  aabb_t operator()(const real_s f) const noexcept
  {
      aabb_t retval;
      retval.upper = f;
      retval.lower = f;
      return retval;
  }
};
// struct distance_calculator
// {
//     __device__
//     real_t operator()(const real_s point, const real_s object) const noexcept
//     {
//         return (point.x - object.x) * (point.x - object.x) +
//                (point.y - object.y) * (point.y - object.y) +
//                (point.z - object.z) * (point.z - object.z);
//     }
// };

int read_pointsC(std::string file_name, std::vector<real_s>& point_cloud)
{
  FILE* fileLAS;

  if((fileLAS = fopen(file_name.c_str(),"r")) == NULL){
    printf("Unable to open file!\n");
    return -1;
  }

  for(auto& p : point_cloud){
    //Obtengo los datos id X Y Z
    if(fscanf(fileLAS, "%lf %lf %lf", &p.x, &p.y, &p.z) < 3){
      printf("Imposible to obtain values\n");
      return -1;
    }
    while(fgetc(fileLAS)!='\n');
  }

  //Ya no necesito mas el fichero
  if(fclose(fileLAS)){
    printf("Cannot close the file\n");
    return -1;
  }

  return 0;
}

__global__ 
void get_morton_code(const real_s* points, const aabb_t BBox, const real_s diffBox, 
                        uint32_t* morton, uint32_t* indices, const size_t N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i<N; i += stride){

    indices[i] = i;

    real_s aux = points[i];

    aux.x -= BBox.lower.x;
    aux.y -= BBox.lower.y;
    aux.z -= BBox.lower.z;
    aux.x /= diffBox.x;
    aux.y /= diffBox.y;
    aux.z /= diffBox.z;

    morton[i] = lbvh::morton_code(aux);

  }

  return;
}

__global__
void check_morton( const uint32_t* m, const uint32_t* idx, uint64_t* morton64, const size_t N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  // uint64_t m64;

  for(int i = index; i<N; i += stride)
  {
    uint64_t m64 = m[i];
    m64 <<= 32;
    m64 |= idx[i];
    morton64[i] = m64;
  }

  return;
}

__global__
void init_leafs(node_t* n, aabb_t* bb, const real_s* p, const uint32_t* idx, const size_t N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i<N; i += stride)
  {
    uint32_t id = idx[i]; // obtengo el índice del punto en esa posición

    real_s auxp = p[id]; // obtengo el punto

    n[i].object_idx = id; // modifico el nodo
    // auxn.object_idx = id; // modifico el nodo

    bb[i] = {auxp,auxp}; // modifico aabb

    // n[i] = auxn;
  }

  return;
}

__global__
void init_leafs2(node_t* n, aabb_t* bb, real_s* p, uint32_t* idx, 
  const size_t nin, const size_t NN, const aabb_t default_aabb)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  node_t auxn;
  auxn.parent_idx = 0xFFFFFFFF;
  auxn.left_idx   = 0xFFFFFFFF;
  auxn.right_idx  = 0xFFFFFFFF;
  auxn.object_idx = 0xFFFFFFFF;

  for(int i = index; i<nin; i += stride)
  {
    n[i] = auxn;
    bb[i] = default_aabb;
  }

  for(int i = index + nin; i<NN; i += stride)
  {
    uint32_t id = idx[i]; // obtengo el índice del punto en esa posición

    real_s auxp = p[id]; // obtengo el punto

    // creo un BB que es solo el punto
    aabb_t auxbb;
    auxbb.upper = auxp;
    auxbb.lower = auxp;

    auxn.object_idx = id; // seteo el índice del nodo

    bb[i] = auxbb;
    n[i] = auxn;
  }

  return;
}

__global__
void init_indices(node_t* n, uint32_t* idx, const size_t N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i<N; i += stride)
  {
    n[i].object_idx = idx[i];
  }

  return;
}


/* Este pequeño código sirve para demostrar que NVCC permite compilar
kernels recursivos */
// __device__ 
// void recursivo(int id)
// {
//   if(id==5){

//   }else{
//     recursivo(id+1);
//   }

//   return;
// }
// __global__
// void launch_recursivo()
// {
//   recursivo(0);
//   return;
// }

// __device__ __forceinline__
// int common_upper_bits(const uint64_t lhs, const uint64_t rhs) noexcept
// {
//     return ::__clzll(lhs ^ rhs);
// }

__device__
inline void swap(uint32_t& left, uint32_t& right)
{
  uint32_t aux = left;
  left = right;
  right = aux;
}

__device__ __forceinline__
uint2 determine_range(const uint64_t* node_code,
        const uint32_t num_leafs, uint32_t idx)
{
  if(idx == 0)
  {
      return make_uint2(0, num_leafs-1);
  }

  // determine direction of the range
  const uint64_t self_code = node_code[idx];
  const int L_delta = ::__clzll(self_code ^ node_code[idx-1]);
  const int R_delta = ::__clzll(self_code ^ node_code[idx+1]);
  const int d = (R_delta > L_delta) ? 1 : -1;

  // Compute upper bound for the length of the range

  // const int delta_min = min(L_delta, R_delta);
  const int delta_min = ::__clzll(self_code ^ node_code[idx-d]);
  int l_max = 2;
  int delta = -1;
  int i_tmp = idx + l_max * d;
  if(0 <= i_tmp && i_tmp < num_leafs)
  {
      delta = ::__clzll(self_code ^ node_code[i_tmp]);
  }
  while(delta > delta_min)
  {
      l_max <<= 1;
      delta = -1;
      i_tmp = idx + l_max * d;
      if(0 <= i_tmp && i_tmp < num_leafs)
      {
          delta = ::__clzll(self_code ^ node_code[i_tmp]);
      }
  }

  // Find the other end by binary search
  uint32_t l = 0;
  // int t = l_max >> 1;
  for(uint32_t t= l_max >> 1; t>0; t>>=1)
  {
    i_tmp = idx + (l + t) * d;
    delta = -1;
    if(0 <= i_tmp && i_tmp < num_leafs)
    {
      if( ::__clzll(self_code ^ node_code[i_tmp]) > delta_min )
        l += t;
    }
  }

  uint32_t jdx = idx + l * d;
  if(d < 0)
  {
      swap(idx, jdx); // make it sure that idx < jdx
  }

#ifdef DEBUG
  printf("L_d: %d, R_d: %d, d: %d, range: [%u,%u]\n",
          L_delta, R_delta, d, idx, jdx);
#endif


  return make_uint2(idx, jdx);
}

// __device__ __inline__ int   flo     (unsigned int v)        { unsigned int r; asm("bfind.u32 %0, %1;" : "=r"(r) : "r"(v)); return r; }
// __device__ __inline__ int   slct    (int a, int b, int c)   { int v; asm("slct.s32.s32 %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }

// __device__ __inline__ int clz(unsigned int hi, unsigned int lo)
// {
//     int a = flo(hi);
//     int b = flo(lo);
//     return slct(31 - a, 63 - b, a);
// }

__device__ __forceinline__
uint2 determine_range2(const uint64_t* node_code,
        const uint32_t num_leafs, uint32_t idx)
{
  if(idx==0) return make_uint2(0, num_leafs-1);

  // Choose direction.
  unsigned int code = node_code[idx];
  int prefix_prev = ::__clzll(code ^ node_code[idx - 1]);
  int prefix_next = ::__clzll(code ^ node_code[idx + 1]);

  int d = (prefix_next > prefix_prev) ? 1 : -1;
  int prefix_min = min(prefix_prev, prefix_next);

  // Find upper bound for length.

  int lmax = 128 >> 2;
  unsigned int probe;
  do
  {
    lmax <<= 2;
    probe = idx + lmax * d;
  }
  while(probe < num_leafs && ::__clzll(code ^ node_code[probe]) > prefix_min);

  // Determine length.

  int l = 0;
  for (int t = lmax >> 1; t > 0; t >>= 1)
  {
    probe = idx + (l + t) * d;
    if (probe < num_leafs && ::__clzll(code ^ node_code[probe]) > prefix_min)
        l += t;
  }
  // int j = idx + l * d;

  uint32_t jdx = idx + l * d;
  if(d < 0)
  {
      swap(idx, jdx); // make it sure that idx < jdx
  }

  return make_uint2(idx, jdx);
}

__device__ __forceinline__
uint32_t find_split(const uint64_t* node_code, const uint32_t num_leafs,
    const uint32_t first, const uint32_t last) noexcept
{
    const uint64_t first_code = node_code[first];
    const uint64_t last_code  = node_code[last];
    if (first_code == last_code)
    {
        return (first + last) >> 1;
    }   
    const int delta_node = ::__clzll(first_code ^ last_code);

    // binary search...
    int split  = first;
    int stride = last - first;
    do
    {
        stride = (stride + 1) >> 1;
        const int middle = split + stride;
        if( middle < last && ::__clzll(first_code ^ node_code[middle]) > delta_node)
          split = middle;

    }
    while(stride > 1);

    return split;
}

__device__ __forceinline__
uint2 build_internal(const uint64_t* node_code,
        const uint32_t num_leafs, uint32_t idx)
{
  uint32_t jdx;
  if(idx != 0)
  {
    // determine direction of the range
    const uint64_t self_code = node_code[idx];
    const int L_delta = ::__clzll(self_code ^ node_code[idx-1]);
    const int R_delta = ::__clzll(self_code ^ node_code[idx+1]);
    const int d = (R_delta > L_delta) ? 1 : -1;

    // Compute upper bound for the length of the range

    const int delta_min = ::__clzll(self_code ^ node_code[idx-d]);
    int l_max = 2;
    int delta = -1;
    int i_tmp = idx + l_max * d;
    if(0 <= i_tmp && i_tmp < num_leafs)
    {
        delta = ::__clzll(self_code ^ node_code[i_tmp]);
    }
    while(delta > delta_min)
    {
        l_max <<= 1;
        delta = -1;
        i_tmp = idx + l_max * d;
        if(0 <= i_tmp && i_tmp < num_leafs)
        {
            delta = ::__clzll(self_code ^ node_code[i_tmp]);
        }
    }

    // Find the other end by binary search
    uint32_t l = 0;
    for(uint32_t t= l_max >> 1; t>0; t>>=1)
    {
      i_tmp = idx + (l + t) * d;
      delta = -1;
      if(0 <= i_tmp && i_tmp < num_leafs)
      {
        if( ::__clzll(self_code ^ node_code[i_tmp]) > delta_min )
          l += t;
      }
    }

    jdx = idx + l * d;
    if(d < 0)
    {
        swap(idx, jdx); // make it sure that idx < jdx
    }
  }
  else
  {
    jdx = num_leafs-1;
  }

#ifdef DEBUG
  printf("L_d: %d, R_d: %d, d: %d, range: [%u,%u]\n",
          L_delta, R_delta, d, idx, jdx);
#endif

  const uint64_t first_code = node_code[idx];
  const uint64_t last_code  = node_code[jdx];
  uint32_t split;
  if (first_code == last_code)
  {
    split = (idx + jdx) >> 1;
  }
  else{
    const uint32_t delta_node = ::__clzll(first_code ^ last_code);

    // binary search...
    split  = idx;
    uint32_t stride = jdx - idx;
    do
    {
        stride = (stride + 1) >> 1;
        const uint32_t middle = split + stride;
        if( middle < jdx && ::__clzll(first_code ^ node_code[middle]) > delta_node)
          split = middle;

    }
    while(stride > 1);
  }

  uint32_t lidx = split;
  uint32_t ridx = split+1;

  if(min(idx, jdx) == lidx)
  {
      lidx += num_leafs-1;
  }
  if(max(idx, jdx) == ridx)
  {
      ridx += num_leafs-1;
  }

  return make_uint2(lidx, ridx);
}

__global__
void init_nodes( node_t* n, const uint64_t* m64, const size_t num_objects )
{
  const uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  // const uint32_t stride = blockDim.x * gridDim.x;

  // const size_t num_internal_nodes = num_objects-1;

  // for(uint32_t i = idx; i<num_internal_nodes; i += stride)
  if(idx < num_objects-1)
  {
    // const uint2 ij  = determine_range(m64, num_objects, i);

    // // find gamma
    // uint32_t lidx = find_split(m64, num_objects, ij.x, ij.y);

    // uint32_t ridx = lidx + 1;

    // if(min(ij.x, ij.y) == lidx)
    // {
    //     lidx += num_internal_nodes;
    // }
    // if(max(ij.x, ij.y) == ridx)
    // {
    //     ridx += num_internal_nodes;
    // }
    // // node_listS[idx] = aux;

    const uint2 ij = build_internal(m64, num_objects, idx);

    n[idx].left_idx = ij.x;
    n[idx].right_idx = ij.y;
    n[ij.x].parent_idx  = idx;
    n[ij.y].parent_idx = idx;

  }

  return;

}
__global__
void init_nodes2( node_t* n, const uint64_t* node_code, const uint32_t num_leafs )
{
  const uint32_t index = threadIdx.x + blockIdx.x * blockDim.x;
  const uint32_t stride = blockDim.x * gridDim.x;

  for(int i=index; i<num_leafs-1; i+=stride)
  {
    
    // const uint2 ij = build_internal(node_code, num_leafs, idx);

    uint32_t idx = i;

    const uint64_t self_code = node_code[idx];
    const int L_delta = (i==0)? FW_S32_MIN : ::__clzll(self_code ^ node_code[idx-1]);
    const int R_delta = ::__clzll(self_code ^ node_code[idx+1]);
    const int d = (R_delta > L_delta) ? 1 : -1;

    // Compute upper bound for the length of the range

    // const int delta_min = ::__clzll(self_code ^ node_code[idx-d]);
    const int delta_min = min(L_delta,R_delta);
    uint32_t l_max = 64;
    int i_tmp = idx + l_max * d;

    do{

      l_max<<=1;
      i_tmp = idx + l_max * d;

    } while( 0 <= i_tmp && i_tmp < num_leafs && delta_min < ::__clzll(self_code ^ node_code[i_tmp]) );


    // Find the other end by binary search
    uint32_t l = 0;
    for(uint32_t t= l_max >> 1; t>0; t>>=1)
    {
      i_tmp = idx + (l + t) * d;
      if( 0 <= i_tmp && i_tmp < num_leafs && delta_min < ::__clzll(self_code ^ node_code[i_tmp]))
        l += t;
    }

    uint32_t jdx = idx + l * d;
    if(d < 0)
    {
        swap(idx, jdx); // make it sure that idx < jdx
    }

    const uint64_t first_code = node_code[idx];
    // const uint64_t last_code  = node_code[jdx];
    const uint32_t prefix_node = ::__clzll(first_code ^ node_code[jdx]);

    // binary search...
    uint32_t split  = idx;
    uint32_t stride = l;

    do
    {
        stride = (stride + 1) >> 1;
        const uint32_t middle = split + stride;
        if( middle < jdx && prefix_node < ::__clzll(first_code ^ node_code[middle]))
          split = middle;

    } while(stride > 1);

    uint32_t lidx = (idx == split) ? (split + num_leafs - 1) : split;
    uint32_t ridx = (jdx == split + 1) ? (split + num_leafs) : (split + 1);

    n[i].left_idx = lidx;
    n[i].right_idx = ridx;
    n[lidx].parent_idx  = i;
    n[ridx].parent_idx = i;
  }

  return;

}

__global__
void init_nodes3( node_t* n, const uint64_t* node_code, const uint32_t num_leafs )
{
  const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;

  if(i < num_leafs-1)
  {
    // const uint2 ij = build_internal(node_code, num_leafs, idx);

    uint32_t idx = i;
    // Choose direction.
    unsigned int code = node_code[idx];
    int prefix_prev = (i==0)? FW_S32_MIN : ::__clzll(code ^ node_code[idx - 1]);
    int prefix_next = ::__clzll(code ^ node_code[idx + 1]);

    int d = (prefix_next > prefix_prev) ? 1 : -1;
    int prefix_min = min(prefix_prev, prefix_next);

    // Find upper bound for length.

    int lmax = 128 >> 2;
    unsigned int probe;
    do
    {
      lmax <<= 2;
      probe = idx + lmax * d;
    }
    while(probe < num_leafs && ::__clzll(code ^ node_code[probe]) > prefix_min);

    // Determine length.

    int l = 0;
    for (int t = lmax >> 1; t > 0; t >>= 1)
    {
      probe = idx + (l + t) * d;
      if (probe < num_leafs && ::__clzll(code ^ node_code[probe]) > prefix_min)
          l += t;
    }
    // int j = idx + l * d;

    uint32_t jdx = idx + l * d;

    int prefix_node = ::__clzll(code ^ node_code[jdx]);

    int s = 0;
    int t = l;
    do
    {
        t = (t + 1) >> 1;
        probe = idx + (s + t) * d;
        if (probe < (unsigned int)num_leafs && prefix_node < ::__clzll(code ^ node_code[probe]))
            s += t;
    }
    while (t > 1);
    int k = idx + s * d + min(d, 0);

    // Output node.

    // int lo = min(idx, jdx);
    // int hi = max(idx, jdx);
  
    uint32_t lidx = (min(idx, jdx) == k) ? (k + num_leafs - 1) : k;
    uint32_t ridx = (max(idx, jdx) == k + 1) ? (k + num_leafs) : (k + 1);

  
    n[i].left_idx = lidx;
    n[i].right_idx = ridx;
    n[lidx].parent_idx  = i;
    n[ridx].parent_idx = i;

  }

  return;

}

__global__
void init_nodes4( node_t* n, const uint64_t* node_code, const uint32_t num_leafs )
{
  const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;

  if(i >= num_leafs-1)
    return;
    
  // const uint2 ij = build_internal(node_code, num_leafs, idx);

  uint32_t idx = i;

  const uint64_t self_code = node_code[idx];
  const int L_delta = (i==0)? FW_S32_MIN : ::__clzll(self_code ^ node_code[idx-1]);
  const int R_delta = ::__clzll(self_code ^ node_code[idx+1]);
  const int d = (R_delta > L_delta) ? 1 : -1;

  // Compute upper bound for the length of the range

  // const int delta_min = ::__clzll(self_code ^ node_code[idx-d]);
  const int delta_min = min(L_delta,R_delta);
  uint32_t l_max = 64;
  int i_tmp = idx + l_max * d;

  do{

    l_max<<=1;
    i_tmp = idx + l_max * d;

  } while( 0 <= i_tmp && i_tmp < num_leafs && delta_min < ::__clzll(self_code ^ node_code[i_tmp]) );


  // Find the other end by binary search
  uint32_t l = 0;
  for(uint32_t t= l_max >> 1; t>0; t>>=1)
  {
    i_tmp = idx + (l + t) * d;
    if( 0 <= i_tmp && i_tmp < num_leafs && delta_min < ::__clzll(self_code ^ node_code[i_tmp]))
      l += t;
  }

  uint32_t jdx = idx + l * d;
  // if(d < 0)
  // {
  //     swap(idx, jdx); // make it sure that idx < jdx
  // }

  // const uint64_t first_code = node_code[idx];
  // const uint64_t last_code  = node_code[jdx];
  const uint32_t prefix_node = ::__clzll(self_code ^ node_code[jdx]);

  // binary search...
  uint32_t split  = 0;
  uint32_t stride = l;

  do
  {
      stride = (stride + 1) >> 1;
      i_tmp = idx + (split + stride) * d;
      if( 0 <= i_tmp && i_tmp < num_leafs && prefix_node < ::__clzll(self_code ^ node_code[i_tmp]))
        split += stride;

  } while(stride > 1);
  
  const uint32_t k = idx + split * d + min(d, 0);

  uint32_t lidx = (min(idx,jdx) == k) ? (k + num_leafs - 1) : k;
  uint32_t ridx = (max(idx,jdx) == k + 1) ? (k + num_leafs) : (k + 1);

  n[i].left_idx = lidx;
  n[i].right_idx = ridx;
  n[lidx].parent_idx  = i;
  n[ridx].parent_idx = i;


  return;

}


__global__
void create_aabb( const node_t* n, aabb_t* bb, uint32_t* flags, const size_t N, const size_t num_nodes)
{
  uint32_t index = N-1 + threadIdx.x + blockIdx.x * blockDim.x;
  uint32_t stride = blockDim.x * gridDim.x;

  for(uint32_t i = index; i < num_nodes; i += stride)
  {
    uint32_t parent = n[i].parent_idx;
    while(parent != 0xFFFFFFFF) // means idx == 0
    {
      const int old = atomicCAS(flags + parent, 0u, 1u);
      if(old == 0)
      {
          // this is the first thread entered here.
          // wait the other thread from the other child node.
          return;
      }
      assert(old == 1);
      // here, the flag has already been 1. it means that this
      // thread is the 2nd thread. merge AABB of both childlen.

      const uint32_t lidx = n[parent].left_idx;
      const uint32_t ridx = n[parent].right_idx;
      const aabb_t lbox = bb[lidx];
      const aabb_t rbox = bb[ridx];
      bb[parent] = lbvh::merge(lbox, rbox);

      // look the next parent...
      parent = n[parent].parent_idx;
    }

  }

  return;
}

__global__
void create_aabb2( const node_t* n, aabb_t* bb, uint32_t* flags, const size_t N, const size_t num_nodes)
{
  uint32_t index = N-1 + threadIdx.x + blockIdx.x * blockDim.x;
  uint32_t stride = blockDim.x * gridDim.x;

  for(uint32_t i = index; i < num_nodes; i += stride)
  {
    uint32_t parent = n[i].parent_idx;

    while(parent != 0xFFFFFFFF && atomicCAS(flags + parent, 0u, 1u)) 
    {
      // here, the flag has already been 1. it means that this
      // thread is the 2nd thread. merge AABB of both childlen.

      const uint32_t lidx = n[parent].left_idx;
      const uint32_t ridx = n[parent].right_idx;
      const aabb_t lbox = bb[lidx];
      const aabb_t rbox = bb[ridx];
      bb[parent] = lbvh::merge(lbox, rbox);

      // look the next parent...
      parent = n[parent].parent_idx;
    }

  }

  return;
}

void create_aabb_cpu( const node_t* n, aabb_t* bb, const uint32_t current)
{
  if(n[current].object_idx != 0xFFFFFFFF)
  {
    return;
  }
  else
  {
    const uint32_t lidx = n[current].left_idx;
    const uint32_t ridx = n[current].right_idx;
    tbb::parallel_invoke([&]() { create_aabb_cpu( n, bb, lidx);}, [&]() { create_aabb_cpu( n, bb, ridx);} );
    // create_aabb_cpu( n, bb, lidx);
    // create_aabb_cpu( n, bb, ridx);
    const aabb_t lbox = bb[lidx];
    const aabb_t rbox = bb[ridx];
    bb[current] = merge(lbox, rbox);
  }
  return;
}

__global__
void init_struct(node_t* n, aabb_t* bb, const size_t NN, const node_t default_node, const aabb_t default_aabb)
{
  uint32_t index = threadIdx.x + blockIdx.x * blockDim.x;
  uint32_t stride = blockDim.x * gridDim.x;

  for(uint32_t i = index; i<NN; i += stride)
  {
    n[i] = default_node;
    bb[i] = default_aabb;
  }
  return;
}

template<typename T, typename L, typename def>
__global__
void init_struct(T* pointer, const L length, const def default_item)
{
  uint32_t index = threadIdx.x + blockIdx.x * blockDim.x;
  uint32_t stride = blockDim.x * gridDim.x;

  for(uint32_t i = index; i<length; i += stride)
  {
    pointer[i] = default_item;
  }
  return;
}


int main(int argc, char* argv[])
{
  std::string inputTXT = (argc > 1)? argv[1] : "data/INAER_2011_Alcoy.xyz";
  std::size_t N;
  aabb_t BBox;

  // const std::size_t N = (argc > 1)? atoi(argv[1]) : 10;

  std::mt19937 mt(123456789);
  // std::uniform_real_distribution<real_t> uni(0.0, 1.0);
  std::uniform_real_distribution<real_t> uni(3000.0, 5000.0);

  if( inputTXT.find("INAER_2011_Alcoy.xyz") != std::string::npos ){
    N = 2772832;
    BBox.lower.x   = 715244.96;
    BBox.lower.y   = 4286623.63;
    BBox.lower.z   = 836.424;
    BBox.upper.x   = 716057.75;
    BBox.upper.y   = 4287447.70;
    BBox.upper.z   = 976.790;

  } else if( inputTXT.find("INAER_2011_Alcoy_Core.xyz") != std::string::npos ){ // Alcoy
    N = 20380212;
    BBox.lower.x = 714947.98;
    BBox.lower.y = 4286501.93;
    BBox.lower.z = 830.381;
    BBox.upper.x = 716361.06;
    BBox.upper.y = 4288406.23;
    BBox.upper.z = 991.516;

  } else if( inputTXT.find("BABCOCK_2017_Arzua_3B.xyz") != std::string::npos ){ //Arzua
    N = 40706503;
    BBox.lower.x = 568000.00;
    BBox.lower.y = 4752320.00;
    BBox.lower.z = 331.620;
    BBox.upper.x = 568999.99;
    BBox.upper.y = 4753319.99;
    BBox.upper.z = 495.630;

  } else if( inputTXT.find("V21_group1_densified_point_cloud.xyz") != std::string::npos ){ //Brion forestal
    N = 42384876;
    BBox.lower.x = 526964.093;
    BBox.lower.y = 4742610.292;
    BBox.lower.z = 38.656;
    BBox.upper.x = 527664.647;
    BBox.upper.y = 4743115.738;
    BBox.upper.z = 112.269;

  } else if( inputTXT.find("V19_group1_densified_point_cloud.xyz") != std::string::npos ){ //Brion urban
    N = 48024480;
    BBox.lower.x = 526955.908;
    BBox.lower.y = 4742586.025;
    BBox.lower.z = 38.150;
    BBox.upper.x = 527686.445;
    BBox.upper.y = 4743124.373;
    BBox.upper.z = 119.833;

  } else {
    printf("No header data!\n");
    exit(-1);
  }

  N = (argc > 2)? static_cast<size_t>(atoi(argv[2])) : N; 

  tbb::global_control c(tbb::global_control::max_allowed_parallelism, 6);

  real_s diffBox;
  diffBox.x = BBox.upper.x - BBox.lower.x;
  diffBox.y = BBox.upper.y - BBox.lower.y;
  diffBox.z = BBox.upper.z - BBox.lower.z;


  std::vector<real_s> ps(N);

  std::cout << N << " points\n";

  if(read_pointsC(inputTXT, ps) < 0){
      printf("Unable to read file!\n");
      exit(-1);
  }

  // for(auto& p : ps)
  // {
  //     p.x = uni(mt);
  //     p.y = uni(mt);
  //     p.z = uni(mt);
  // }


  real_s*    point_cloud = NULL;
  uint32_t*  morton = NULL;
  uint64_t*  morton64 = NULL;
  uint32_t*  morton_out = NULL;
  uint32_t*  indices = NULL;
  uint32_t*  indices_out = NULL;
  void*      d_temp_storage = NULL;

  node_t* node_list = NULL;
  node_t* aux_node_list = NULL;
  aabb_t* aabb_list = NULL;
  aabb_t* aux_aabb_list = NULL;

  uint32_t* flags;

  const size_t num_objects = N;
  const size_t num_internal_nodes = num_objects - 1;
  const size_t num_nodes = 2*num_objects - 1; /*Numero de Nodos*/ 

  int deviceId;
  int numberOfSMs;
  const size_t size = num_objects * sizeof(real_s);
  const size_t size_morton = num_objects * sizeof(uint32_t);
  const size_t size_morton64 = num_objects * sizeof(uint64_t);
  size_t temp_storage_bytes = 0;
  
  const size_t size_nodes = num_nodes * sizeof(node_t);
  const size_t size_aabbs = num_nodes * sizeof(aabb_t);

  hipError_t mortonError;
  hipError_t asyncErr;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
  printf("Device ID: %d\tNumber of SMs: %d\n", deviceId, numberOfSMs);

  size_t threadsPerBlock = 256;
  size_t numberOfBlocks = 32 * numberOfSMs;

  uint32_t nBlocks_aabb_func = (uint32_t)(N/threadsPerBlock) + 1;

  hipMallocManaged(&point_cloud, size);
  hipMallocManaged(&morton, size_morton);
  hipMallocManaged(&morton64, size_morton64);
  hipMallocManaged(&morton_out, size_morton);
  hipMallocManaged(&indices, size_morton);
  hipMallocManaged(&indices_out, size_morton);

  hipMallocManaged(&node_list, size_nodes);
  aux_node_list = &node_list[num_internal_nodes];
  hipMallocManaged(&aabb_list, size_aabbs);
  aux_aabb_list = &aabb_list[num_internal_nodes];

  hipMallocManaged(&flags, N*sizeof(uint32_t));

  hipMemPrefetchAsync(point_cloud, size, hipCpuDeviceId);

  int i = 0;
  for(auto& p : ps){
    point_cloud[i].x = p.x;
    point_cloud[i].y = p.y;
    point_cloud[i].z = p.z;
    i++;
  }

  node_t default_node;
  default_node.parent_idx = 0xFFFFFFFF;
  default_node.left_idx   = 0xFFFFFFFF;
  default_node.right_idx  = 0xFFFFFFFF;
  default_node.object_idx = 0xFFFFFFFF;


  const auto inf = std::numeric_limits<double>::infinity();
  aabb_t default_aabb;
  default_aabb.upper.x = -inf; default_aabb.lower.x = inf;
  default_aabb.upper.y = -inf; default_aabb.lower.y = inf;
  default_aabb.upper.z = -inf; default_aabb.lower.z = inf;

  hipMemPrefetchAsync(node_list, size_nodes, deviceId);
  hipMemPrefetchAsync(aabb_list, size_aabbs, deviceId);

  init_struct<<<numberOfBlocks, threadsPerBlock>>>( node_list, aabb_list, num_nodes, default_node, default_aabb);
  // init_struct<<<numberOfBlocks, threadsPerBlock>>>( aabb_list, num_internal_nodes, default_aabb);
  // init_struct<<<numberOfBlocks, threadsPerBlock>>>( node_list, num_internal_nodes, default_node);

  mortonError = hipGetLastError();
  if(mortonError != hipSuccess) printf("Error INIT NODES y ABBBs: %s\n", hipGetErrorString(mortonError));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error INIT NODES y ABBBs sync: %s\n", hipGetErrorString(asyncErr));


  hipMemPrefetchAsync(point_cloud, size, deviceId);
  hipMemPrefetchAsync(morton, size_morton, deviceId);
  hipMemPrefetchAsync(indices, size_morton, deviceId);

  std::chrono::time_point<tempo_t> i_start = tempo_t::now();

  get_morton_code<<<numberOfBlocks, threadsPerBlock>>>( point_cloud, BBox, diffBox, morton, indices, num_objects);

  mortonError = hipGetLastError();
  if(mortonError != hipSuccess) printf("Error MORTON: %s\n", hipGetErrorString(mortonError));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error MORTON sync: %s\n", hipGetErrorString(asyncErr));

  double mytime = cast_t(tempo_t::now() - i_start).count();
  std::cout << "  MORTON time elapsed: " << mytime << " ms\n";
  double totaltime = mytime;

#ifdef DEBUG
  for(int i = 0; i<N; i++){
    std::cout << morton[i] << " ";
  }
  std::cout << "\n";
  for(int i = 0; i<N; i++){
    std::cout << indices[i] << " ";
  }
  std::cout << "\n";
#endif

  // hipMemPrefetchAsync(morton, size_morton, deviceId);
  hipMemPrefetchAsync(morton_out, size_morton, deviceId);
  // hipMemPrefetchAsync(indices, size_morton, deviceId);
  hipMemPrefetchAsync(indices_out, size_morton, deviceId);


  i_start = tempo_t::now();

  /* Determine temporary device storage requirements; segun las especificaciones
  si el puntero temporal apunta a NULL, se modifica "temp_storage_bytes" con el
  tamaño de memoria temporal requerida */

  hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
      morton, morton_out, indices, indices_out, N);
  /* Allocate temporary storage */
  hipMallocManaged(&d_temp_storage, temp_storage_bytes);
  hipMemPrefetchAsync(d_temp_storage, temp_storage_bytes, deviceId);
  
  /* solo ordeno los índices */
  hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
      morton, morton_out, indices, indices_out, N);

  mortonError = hipGetLastError();
  if(mortonError != hipSuccess) printf("Error SORT: %s\n", hipGetErrorString(mortonError));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error SORT sync: %s\n", hipGetErrorString(asyncErr));

  mytime = cast_t(tempo_t::now() - i_start).count();
  std::cout << "  SORT time elapsed: " << mytime << " ms\n";
  totaltime += mytime;

#ifdef DEBUG
  for(int i = 0; i<N; i++){
    std::cout << morton_out[i] << " ";
  }
  std::cout << "\n";
  for(int i = 0; i<N; i++){
    std::cout << indices_out[i] << " ";
  }
  std::cout << "\n";
#endif

  i_start = tempo_t::now();

  hipMemPrefetchAsync(morton64, size_morton64, deviceId);

  check_morton<<<numberOfBlocks, threadsPerBlock>>>( morton_out, indices_out, morton64, num_objects );

  mortonError = hipGetLastError();
  if(mortonError != hipSuccess) printf("Error CHECK: %s\n", hipGetErrorString(mortonError));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error CHECK sync: %s\n", hipGetErrorString(asyncErr));

  mytime = cast_t(tempo_t::now() - i_start).count();
  std::cout << "  CHECK time elapsed: " << mytime << " ms\n";
  totaltime += mytime;

#ifdef DEBUG
  for(int i = 0; i<N; i++){
    std::cout << morton64[i] << " ";
  }
  std::cout << "\n";
  for(int i = 0; i<N; i++){
    std::cout << indices_out[i] << " ";
  }
  std::cout << "\n";
#endif

  // i_start = tempo_t::now();

  // hipMemset( node_list, 0xFFFFFFFF, size_nodes );

  // mytime = cast_t(tempo_t::now() - i_start).count();
  // std::cout << "  MemSet time elapsed: " << mytime << " ms\n";

  i_start = tempo_t::now();

  // hipMemsetAsync( node_list, 0xFFFFFFFF, size_nodes*0.5 , (hipStream_t)1);

  hipMemPrefetchAsync(aux_node_list, num_objects*sizeof(node_t), deviceId);
  hipMemPrefetchAsync(aux_aabb_list, num_objects*sizeof(aabb_t), deviceId);

  mortonError = hipGetLastError();
  if(mortonError != hipSuccess) printf("Error LEAFS memory: %s\n", hipGetErrorString(mortonError));

  init_leafs<<<numberOfBlocks, threadsPerBlock>>>( aux_node_list, aux_aabb_list, point_cloud, indices_out, num_objects );

  // init_leafs<<<numberOfBlocks, threadsPerBlock>>>( aux_node_list, indices, N );

  // init_leafs2<<<numberOfBlocks, threadsPerBlock>>>( node_list, aabb_list, point_cloud, indices, num_internal_nodes, num_nodes );

  // init_indices<<<numberOfBlocks, threadsPerBlock>>>( aux_node_list, indices, N );

  mortonError = hipGetLastError();
  if(mortonError != hipSuccess) printf("Error LEAFS: %s\n", hipGetErrorString(mortonError));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error LEAFS sync: %s\n", hipGetErrorString(asyncErr));

  mytime = cast_t(tempo_t::now() - i_start).count();
  std::cout << "  LEAFS and AABBs time elapsed: " << mytime << " ms\n";
  totaltime += mytime;

#ifdef DEBUG
  for(int i = 0; i<num_nodes; i++){
      std::cout << node_list[i].parent_idx << ", " << node_list[i].left_idx << ", ";
      std::cout << node_list[i].right_idx << ", " << node_list[i].object_idx << "\n";
  }
  std::cout << std::endl;

  for(int i = 0; i<num_nodes; i++){
      std::cout << aabb_list[i].upper.x << "," << aabb_list[i].upper.y << "," << aabb_list[i].upper.z << " ";
      std::cout << aabb_list[i].lower.x << "," << aabb_list[i].lower.y << "," << aabb_list[i].lower.z << "\n";
  }
  std::cout << std::endl;
#endif

  i_start = tempo_t::now();

  hipMemPrefetchAsync(node_list, size_nodes, deviceId);
  hipMemPrefetchAsync(morton64, size_morton64, deviceId);

  // init_nodes<<<nBlocks_aabb_func, threadsPerBlock>>>( node_list, morton64, num_objects );
  init_nodes2<<<numberOfBlocks, threadsPerBlock>>>( node_list, morton64, num_objects );
  // init_nodes3<<<nBlocks_aabb_func, threadsPerBlock>>>( node_list, morton64, num_objects );
  // init_nodes4<<<nBlocks_aabb_func, threadsPerBlock>>>( node_list, morton64, num_objects );
  
  // thrust::for_each(thrust::device,
  //     thrust::make_counting_iterator<uint32_t>(0),
  //     thrust::make_counting_iterator<uint32_t>(num_internal_nodes),
  //     [node_list, morton64, num_objects] __device__ (const uint32_t idx)
  //     {
  //       // node_t aux;

  //       const uint2 ij  = lbvh::detail::determine_range(morton64, num_objects, idx);
  //       // const uint32_t gamma = lbvh::detail::find_split(morton64, num_objects, ij.x, ij.y);
  //       uint32_t left_idx = lbvh::detail::find_split(morton64, num_objects, ij.x, ij.y);

  //       // aux.object_idx = 0xFFFFFFFF;

  //       // node_list[idx].left_idx  = gamma;
  //       // node_list[idx].right_idx = gamma + 1;
  //       uint32_t right_idx = left_idx + 1;

  //       if(thrust::min(ij.x, ij.y) == left_idx)
  //       {
  //           left_idx += num_objects - 1;
  //       }
  //       if(thrust::max(ij.x, ij.y) == right_idx)
  //       {
  //           right_idx += num_objects - 1;
  //       }
  //       // node_list[idx] = aux;

  //       node_list[idx].left_idx = left_idx;
  //       node_list[idx].right_idx = right_idx;
  //       node_list[left_idx].parent_idx  = idx;
  //       node_list[right_idx].parent_idx = idx;
  //       return;
  //     });

  mortonError = hipGetLastError();
  if(mortonError != hipSuccess) printf("Error NODES: %s\n", hipGetErrorString(mortonError));

  // asyncErr = hipDeviceSynchronize();
  // if(asyncErr != hipSuccess) printf("Error NODES sync: %s\n", hipGetErrorString(asyncErr));

  mytime = cast_t(tempo_t::now() - i_start).count();
  std::cout << "  INTERNAL NODES time elapsed: " << mytime << " ms\n";
  totaltime += mytime;

#ifdef DEBUG
  for(int i = 0; i<num_nodes; i++){
      std::cout << node_list[i].parent_idx << ", " << node_list[i].left_idx << ", ";
      std::cout << node_list[i].right_idx << ", " << node_list[i].object_idx << "\n";
  }
  std::cout << std::endl;
#endif


  i_start = tempo_t::now();

  hipMemPrefetchAsync(flags, num_objects*sizeof(uint32_t), deviceId);
  hipMemPrefetchAsync(aabb_list, size_aabbs, deviceId);

  // create_aabb<<<nBlocks_aabb_func, threadsPerBlock>>>( node_list, aabb_list, flags, num_objects, num_nodes );
  create_aabb2<<<numberOfBlocks, threadsPerBlock>>>( node_list, aabb_list, flags, num_objects, num_nodes );
  // create_aabb_cpu(node_list, aabb_list, 0);

  // thrust::for_each(thrust::device,
  //     thrust::make_counting_iterator<uint32_t>(num_internal_nodes),
  //     thrust::make_counting_iterator<uint32_t>(num_nodes),
  //     [node_list, aabb_list, flags] __device__ (const uint32_t idx)
  //     {
  //         uint32_t parent = node_list[idx].parent_idx;
  //         while(parent != 0xFFFFFFFF) // means idx == 0
  //         {
  //             const int old = atomicCAS(flags + parent, 0, 1);
  //             if(old == 0)
  //             {
  //                 // this is the first thread entered here.
  //                 // wait the other thread from the other child node.
  //                 return;
  //             }
  //             assert(old == 1);
  //             // here, the flag has already been 1. it means that this
  //             // thread is the 2nd thread. merge AABB of both childlen.

  //             const uint32_t lidx = node_list[parent].left_idx;
  //             const uint32_t ridx = node_list[parent].right_idx;
  //             const aabb_t lbox = aabb_list[lidx];
  //             const aabb_t rbox = aabb_list[ridx];
  //             aabb_list[parent] = lbvh::merge(lbox, rbox);

  //             // look the next parent...
  //             parent = node_list[parent].parent_idx;
  //         }
  //         return;
  //     });

  mortonError = hipGetLastError();
  if(mortonError != hipSuccess) printf("Error AABBs: %s\n", hipGetErrorString(mortonError));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error AABBs sync: %s\n", hipGetErrorString(asyncErr));

  mytime = cast_t(tempo_t::now() - i_start).count();
  std::cout << "  Create AABB time elapsed: " << mytime << " ms\n";
  totaltime += mytime;
  std::cout << "  CREATION takes: " << totaltime << " ms\n";

#ifdef DEBUG

  for(int i = 0; i<num_nodes; i++){
      std::cout << aabb_list[i].upper.x << "," << aabb_list[i].upper.y << "," << aabb_list[i].upper.z << " ";
      std::cout << aabb_list[i].lower.x << "," << aabb_list[i].lower.y << "," << aabb_list[i].lower.z << "\n";
  }
  std::cout << std::endl;
#endif


#ifdef CHECK
  std::cout << "\n\n";
  lbvh::bvh<real_t, real_s, aabb_getter> bvh(ps.begin(), ps.end(), true);
  std::cout << "\n\n";


  auto bvh_nodes = bvh.nodes_host();
  auto bvh_aabbs = bvh.aabbs_host();
  auto bvh_morton = bvh.morton_host();
  auto bvh_morton64 = bvh.morton64_host();
  auto bvh_indices = bvh.indices_host();

  int error=0;
  for(int i=0; i<num_nodes; i++){
      // std::cout << bvh_nodes[i].parent_idx << ", " << node_list[i].parent_idx << "\n";
      if(bvh_nodes[i].parent_idx != node_list[i].parent_idx ||
          bvh_nodes[i].left_idx != node_list[i].left_idx ||
          bvh_nodes[i].right_idx != node_list[i].right_idx ||
          bvh_nodes[i].object_idx != node_list[i].object_idx )
        error++;
  }

  if(error)
    std::cout << "There are DIFFERENCES in NODES: " << error << std::endl;
  else
    std::cout << "NODES are OK!\n";
  
  error = 0;
  for(int i=0; i<num_nodes; i++){
      if( fabs(bvh_aabbs[i].upper.x - aabb_list[i].upper.x) > TOL ||
          fabs(bvh_aabbs[i].upper.y - aabb_list[i].upper.y) > TOL ||
          fabs(bvh_aabbs[i].upper.z - aabb_list[i].upper.z) > TOL ||
          fabs(bvh_aabbs[i].lower.x - aabb_list[i].lower.x) > TOL ||
          fabs(bvh_aabbs[i].lower.y - aabb_list[i].lower.y) > TOL ||
          fabs(bvh_aabbs[i].lower.z - aabb_list[i].lower.z) > TOL )
        error++;
  }

  if(error)
    std::cout << "There are DIFFERENCES in AABBS: " << error << std::endl;
  else
    std::cout << "AABBs are OK!\n";


  error=0;
  for(int i=0; i<num_objects; i++){
      if(bvh_morton[i] != morton_out[i])
        error++;
  }

  if(error)
    std::cout << "There are DIFFERENCES in MORTON: " << error << std::endl;
  else
    std::cout << "MORTONs are OK!\n";

  error=0;
  for(int i=0; i<num_objects; i++){
      if(bvh_indices[i] != indices_out[i])
        error++;
  }

  if(error)
    std::cout << "There are DIFFERENCES in INDICES: " << error << std::endl;
  else
    std::cout << "INDICES are OK!\n";


  error=0;
  for(int i=0; i<num_objects; i++){
      if(bvh_morton64[i] != morton64[i])
        error++;
  }

  if(error)
    std::cout << "There are DIFFERENCES in MORTON64: " << error << std::endl;
  else
    std::cout << "MORTON64s are OK!\n";


  bvh.clear();

#endif

  hipFree(point_cloud);
  hipFree(morton);
  hipFree(morton64);
  hipFree(morton_out);
  hipFree(indices);
  hipFree(indices_out);
  hipFree(d_temp_storage);

  hipFree(node_list);
  aux_node_list = NULL;
  hipFree(aabb_list);
  aux_aabb_list = NULL;

  hipFree(flags);

  // mytime = cast_t(tempo_t::now() - i_start).count();
  // double cpu_tree_time = mytime/1e3;

  // std::cout << "  CREATION time elapsed: " << cpu_tree_time << " s\n";

  // const auto bvh_dev = bvh.get_device_repr();

  // std::cout << "testing query_device:overlap ...\n";
  // thrust::for_each(thrust::device,
  //     thrust::make_counting_iterator<std::size_t>(0),
  //     thrust::make_counting_iterator<std::size_t>(N),
  //     [bvh_dev] __device__ (std::size_t idx) {
  //         unsigned int buffer[10];
  //         const auto self = bvh_dev.objects[idx];
  //         const real_t  dr = 0.1f;
  //         for(std::size_t i=1; i<10; ++i)
  //         {
  //             for(unsigned int j=0; j<10; ++j)
  //             {
  //                 buffer[j] = 0xFFFFFFFF;
  //             }
  //             const real_t r = dr * i;
  //             lbvh::aabb<real_t> query_box;
  //             query_box.lower = make_real_s(self.x-r, self.y-r, self.z-r, 0);
  //             query_box.upper = make_real_s(self.x+r, self.y+r, self.z+r, 0);
  //             const auto num_found = lbvh::query_device(
  //                     bvh_dev, lbvh::overlaps(query_box), buffer, 10);

  //             for(unsigned int j=0; j<10; ++j)
  //             {
  //                 const auto jdx    = buffer[j];
  //                 if(j >= num_found)
  //                 {
  //                     assert(jdx == 0xFFFFFFFF);
  //                     continue;
  //                 }
  //                 else
  //                 {
  //                     assert(jdx != 0xFFFFFFFF);
  //                     assert(jdx < bvh_dev.num_objects);
  //                 }
  //                 const auto other  = bvh_dev.objects[jdx];
  //                 assert(fabsf(self.x - other.x) < r); // check coordinates
  //                 assert(fabsf(self.y - other.y) < r); // are in the range
  //                 assert(fabsf(self.z - other.z) < r); // of query box
  //             }
  //         }
  //         return ;
  //     });

  // std::cout << "testing query_device:nearest_neighbor ...\n";
  // thrust::for_each(thrust::device,
  //     thrust::make_counting_iterator<unsigned int>(0),
  //     thrust::make_counting_iterator<unsigned int>(N),
  //     [bvh_dev] __device__ (const unsigned int idx) {
  //         const auto self = bvh_dev.objects[idx];
  //         const auto nest = lbvh::query_device(bvh_dev, lbvh::nearest(self),
  //                                              distance_calculator());
  //         assert(nest.first != 0xFFFFFFFF);
  //         const auto other   = bvh_dev.objects[nest.first];
  //         // of course, the nearest object is itself.
  //         assert(nest.second == 0.0f);
  //         assert(self.x == other.x);
  //         assert(self.y == other.y);
  //         assert(self.z == other.z);
  //         return ;
  //    });

  // thrust::device_vector<real_s> random_points(N);
  // thrust::transform(
  //     thrust::make_counting_iterator<unsigned int>(0),
  //     thrust::make_counting_iterator<unsigned int>(N),
  //     random_points.begin(), [] __device__(const unsigned int idx) {
  //         thrust::default_random_engine rand;
  //         thrust::uniform_real_distribution<real_t> uni(0.0f, 1.0f);
  //         rand.discard(idx);
  //         const real_t x = uni(rand);
  //         const real_t y = uni(rand);
  //         const real_t z = uni(rand);
  //         return make_real_s(x, y, z, 0);
  //     });

  // thrust::for_each(random_points.begin(), random_points.end(),
  //     [bvh_dev] __device__ (const real_s pos) {
  //         const auto calc = distance_calculator();
  //         const auto nest = lbvh::query_device(bvh_dev, lbvh::nearest(pos), calc);
  //         assert(nest.first != 0xFFFFFFFF);

  //         for(unsigned int i=0; i<bvh_dev.num_objects; ++i)
  //         {
  //             const auto dist = calc(bvh_dev.objects[i], pos);
  //             if(i == nest.first)
  //             {
  //                 assert(dist == nest.second);
  //             }
  //             else
  //             {
  //                 assert(dist >= nest.second);
  //             }
  //         }
  //         return ;
  //     });

  // std::cout << "testing query_host:overlap ...\n";
  // {
  //     for(std::size_t i=0; i<10; ++i)
  //     {
  //         const auto self = bvh.objects_host()[i];
  //         const real_t dr = 0.1f;
  //         for(unsigned int cnt=1; cnt<10; ++cnt)
  //         {
  //             const real_t r = dr * cnt;
  //             lbvh::aabb<real_t> query_box;
  //             query_box.lower = make_real_s(self.x-r, self.y-r, self.z-r, 0);
  //             query_box.upper = make_real_s(self.x+r, self.y+r, self.z+r, 0);

  //             std::vector<std::size_t> buffer;
  //             const auto num_found = lbvh::query_host(bvh,
  //                     lbvh::overlaps(query_box), std::back_inserter(buffer));

  //             for(unsigned int jdx : buffer)
  //             {
  //                 assert(jdx < bvh.objects_host().size());

  //                 const auto other  = bvh.objects_host()[jdx];
  //                 assert(fabsf(self.x - other.x) < r); // check coordinates
  //                 assert(fabsf(self.y - other.y) < r); // are in the range
  //                 assert(fabsf(self.z - other.z) < r); // of query box
  //             }
  //         }
  //     }
  // }

  return 0;
}
