#include <iostream>

#include <bintree/cuda/bintree_builder.h>

void bintree_test(std::string inputTXT, const uint32_t chunkDim, float& dtime)
{
	aabb_t BBox;
	uint32_t N;

	readHeader(inputTXT, BBox, N);
  
	real_v diffBox;
	diffBox.x = BBox.upper.x - BBox.lower.x;
	diffBox.y = BBox.upper.y - BBox.lower.y;  
  
	point_t*   point_cloud = NULL;
	uint32_t*  morton = NULL;
	uint32_t*  morton_out = NULL;
	uint32_t*  indices = NULL;
	uint32_t*  indices_out = NULL;
	void*      d_temp_storage = NULL;
	size_t temp_storage_bytes = 0;
	  
	uint64_t* morton64S = NULL;
	bintree_node* node_listS = NULL;
	aabb_t* aabb_listS = NULL;
	uint32_t* flagsS = NULL;
    
	int deviceId;
	int numberOfSMs;

	hipGetDevice(&deviceId);
	hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
	// printf("Device ID: %d\tNumber of SMs: %d\n", deviceId, numberOfSMs);
	
	const uint32_t num_objects = N;
	/* reduced number of objects */
	const uint32_t r_num_objects = (uint32_t)((num_objects-1)/chunkDim) + 1u;
	const uint32_t r_num_internal_nodes = r_num_objects - 1u;
	const uint32_t r_num_nodes = 2*r_num_objects - 1u;

	hipError_t lastError;
	// hipError_t asyncErr;
  
	const uint32_t threadsPerBlock = 256;
	const uint32_t numberOfBlocks = 32*numberOfSMs;

	const uint32_t nBlocks_node_func = (uint32_t)((r_num_internal_nodes-1)/threadsPerBlock) + 1;
	const uint32_t nBlocks_aabb_func = (uint32_t)((r_num_objects-1)/threadsPerBlock) + 1;
  
	hipMallocManaged(&point_cloud, num_objects*sizeof(point_t));
	hipMallocManaged(&morton, num_objects*sizeof(uint32_t));
	hipMallocManaged(&morton_out, num_objects*sizeof(uint32_t));
	hipMallocManaged(&indices, num_objects*sizeof(uint32_t));
	hipMallocManaged(&indices_out, num_objects*sizeof(uint32_t));
  
	hipMallocManaged(&morton64S, r_num_objects*sizeof(uint64_t));
	hipMallocManaged(&node_listS, r_num_nodes*sizeof(bintree_node));
	hipMallocManaged(&aabb_listS, r_num_nodes*sizeof(aabb_t));
	hipMallocManaged(&flagsS, r_num_internal_nodes*sizeof(uint32_t));

	hipMemPrefetchAsync(node_listS, r_num_nodes*sizeof(bintree_node), deviceId);
	hipMemPrefetchAsync(aabb_listS, r_num_nodes*sizeof(aabb_t), deviceId);
	hipMemPrefetchAsync(morton, num_objects*sizeof(uint32_t), deviceId);
	hipMemPrefetchAsync(indices, num_objects*sizeof(uint32_t), deviceId);
	hipMemPrefetchAsync(morton_out, num_objects*sizeof(uint32_t), deviceId);
	hipMemPrefetchAsync(indices_out, num_objects*sizeof(uint32_t), deviceId);
	hipMemPrefetchAsync(morton64S, r_num_objects*sizeof(uint64_t), deviceId);
	hipMemPrefetchAsync(flagsS, r_num_internal_nodes*sizeof(uint32_t), deviceId);
  
	hipMemPrefetchAsync(point_cloud, num_objects*sizeof(point_t), hipCpuDeviceId);

	if(read_pointsC(inputTXT, point_cloud, num_objects) < 0){
		printf("Unable to read file!\n");
		exit(-1);
	}

	hipMemPrefetchAsync(point_cloud, num_objects*sizeof(point_t), deviceId);
  
	bintree_builder::init_struct<<<numberOfBlocks, threadsPerBlock>>>( node_listS, aabb_listS, r_num_nodes, default_node, default_aabb);

	lastError = hipGetLastError();
	if(lastError != hipSuccess) printf("Error INIT NODES y ABBBs: %s\n", hipGetErrorString(lastError));
	hipDeviceSynchronize();

    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );

	hipEventRecord( start, 0 );
	  
	bintree_builder::get_morton_code<<<numberOfBlocks, threadsPerBlock>>>( point_cloud, BBox, diffBox, morton, indices, num_objects);
	
	lastError = hipGetLastError();
	if(lastError != hipSuccess) printf("Error MORTON: %s\n", hipGetErrorString(lastError));
	hipDeviceSynchronize();

	/* Determine temporary device storage requirements */
	hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
		morton, morton_out, indices, indices_out, N);

	/* Allocate temporary storage */
	hipMallocManaged(&d_temp_storage, temp_storage_bytes);
	hipMemPrefetchAsync(d_temp_storage, temp_storage_bytes, deviceId);
	
	/* Sort indices */
	hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
		morton, morton_out, indices, indices_out, N);
	
	lastError = hipGetLastError();
	if(lastError != hipSuccess) printf("Error SORT: %s\n", hipGetErrorString(lastError));
	hipDeviceSynchronize();

	bintree_builder::init_leafs_size2<<<numberOfBlocks, threadsPerBlock>>>( 
										&node_listS[r_num_internal_nodes], 
										&aabb_listS[r_num_internal_nodes], 
										point_cloud, 
										indices_out,
										morton64S, 
										num_objects,
										r_num_objects,
										chunkDim,
										default_aabb,
										BBox,
										diffBox );

	lastError = hipGetLastError();
	if(lastError != hipSuccess) printf("Error LEAFS: %s\n", hipGetErrorString(lastError));
	hipDeviceSynchronize();
																		  
	bintree_builder::init_nodes2<<<nBlocks_node_func, threadsPerBlock>>>( 
										node_listS, 
										morton64S, 
										r_num_objects );

	lastError = hipGetLastError();
	if(lastError != hipSuccess) printf("Error NODES: %s\n", hipGetErrorString(lastError));
	hipDeviceSynchronize();
									
	bintree_builder::create_aabb_size<<<nBlocks_aabb_func, threadsPerBlock>>>( 
										node_listS, 
										aabb_listS, 
										flagsS,
										point_cloud, 
										r_num_internal_nodes, 
										r_num_nodes );
	
	
	lastError = hipGetLastError();
	if(lastError != hipSuccess) printf("Error AABBs: %s\n", hipGetErrorString(lastError));
	hipDeviceSynchronize();
							
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &dtime, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );

	hipFree(point_cloud);
	hipFree(morton);
	hipFree(morton_out);
	hipFree(indices);
	hipFree(indices_out);
	hipFree(d_temp_storage);
  
	hipFree(morton64S);
	hipFree(node_listS);
	hipFree(aabb_listS);
	hipFree(flagsS);
	  
	return;
}

int main(int argc, char* argv[])
{

	const std::string inputTXT = (argc > 1)? argv[1] : "../data/INAER_2011_Alcoy.xyz";
	const int chunkDim = (argc > 2)? atoi(argv[2]) : 256;

	float time = 0.0;
	// int nreps = 10;
	// for(int rep=0; rep<nreps; rep++)
	// {
		float dtime=0.0;
		bintree_test(inputTXT, chunkDim, dtime);
		time += dtime;
	// }

	std::cout << " CREATION takes: " << time << " ms\n";

	return 0;
}